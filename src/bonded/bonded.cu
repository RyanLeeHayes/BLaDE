#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "bonded.h"
#include "main/defines.h"
#include "system/system.h"
#include "system/state.h"
#include "msld/msld.h"
#include "run/run.h"
#include "system/potential.h"

#include "main/real3.h"

/*
// In case we need global variables to save time uploading arguments
__device__ int N_bond;
__device__ struct_bondparms *bondparms;
__device__ struct_bondblock bondblock;
__device__ real *sp_box;
__device__ real *sp_Gt;

void upload_bonded_d(
  int N_b,struct_bondparms* h_bondparms,struct_bondblock* h_bondblock,
  struct_atoms h_at,real* h_box,real* h_Gt)
{
  hipMemcpyToSymbol(HIP_SYMBOL(N_bond), &N_b, sizeof(int), size_t(0),hipMemcpyHostToDevice);
  if (N_b) {
    hipMemcpyToSymbol(HIP_SYMBOL(bondparms), &h_bondparms, sizeof(struct_bondparms*), size_t(0),hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(bondblock), &h_bondblock[0], sizeof(struct_bondblock), size_t(0),hipMemcpyHostToDevice);
  }
  hipMemcpyToSymbol(HIP_SYMBOL(sp_at), &h_at, sizeof(struct_atoms), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(sp_box), &h_box, sizeof(real*), size_t(0),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(sp_Gt), &h_Gt, sizeof(real*), size_t(0),hipMemcpyHostToDevice);
}
*/



// getforce_bond_kernel<<<(N+BLBO-1)/BLBO,BLBO,0,p->bondedStream>>>(N,p->bonds,s->position_d,s->force_d,s->box,m->lambda_d,m->lambdaForce_d,NULL);
__global__ void getforce_bond_kernel(int bondCount,struct BondPotential *bonds,real3 *position,real3 *force,real3 box,real *lambda,real *lambdaForce,real *energy)
{
// NYI - maybe energy should be a double
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int ii,jj;
  real r;
  real3 dr;
  BondPotential bp;
  real fbond;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,xj;
  int b[2];
  real l[2]={1,1};
  
  if (i<bondCount) {
    // Geometry
    bp=bonds[i];
    ii=bp.idx[0];
    jj=bp.idx[1];
    xi=position[ii];
    xj=position[jj];
#warning "Unprotected division"
    dr=real3_subpbc(xi,xj,box);
#warning "Unprotected sqrt"
    r=real3_mag(dr);
    
    // Scaling
    b[0]=0xFFFF & bp.siteBlock[0];
    b[1]=0xFFFF & bp.siteBlock[1];
    if (b[0]) {
      l[0]=lambda[b[0]];
      if (b[1]) {
        l[1]=lambda[b[1]];
      }
    }

    // interaction
    fbond=bp.kb*(r-bp.b0);
    if (b[0] || energy) {
      lEnergy=0.5*bp.kb*(r-bp.b0)*(r-bp.b0);
    }

    // Lambda force
    if (b[0]) {
      realAtomicAdd(&lambdaForce[b[0]],l[1]*lEnergy);
      if (b[1]) {
        realAtomicAdd(&lambdaForce[b[1]],l[0]*lEnergy);
      }
    }

    // Spatial force
#warning "division in kernel"
    at_real3_scaleinc(&force[ii], fbond/r,dr);
    at_real3_scaleinc(&force[jj],-fbond/r,dr);
  }

  // Energy, if requested
  if (energy) {
    lEnergy*=l[0]*l[1];
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,8);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,16);
    __syncthreads();
    if ((0x1F & threadIdx.x)==0) {
      sEnergy[threadIdx.x>>5]=lEnergy;
    }
    __syncthreads();
    if (threadIdx.x < (blockDim.x>>5)) {
      lEnergy=sEnergy[threadIdx.x];
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    }
    if (threadIdx.x==0) {
      realAtomicAdd(energy,lEnergy);
    }
  }
}

void getforce_bond(System *system,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Msld *m=system->msld;
  int N=p->bondCount;
  int shMem=0;
  real *pEnergy=NULL;


  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebond;
  }

  getforce_bond_kernel<<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->bonds_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
}



// getforce_angle_kernel<<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->angles_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
__global__ void getforce_angle_kernel(int angleCount,struct AnglePotential *angles,real3 *position,real3 *force,real3 box,real *lambda,real *lambdaForce,real *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int ii,jj,kk;
  AnglePotential ap;
  real3 drij,drkj;
  real t;
  real dotp, mcrop;
  real3 crop;
  real3 fi,fj,fk;
  real fangle;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi, xj, xk;
  int b[2];
  real l[2]={1,1};

  if (i<angleCount) {
    // Geometry
    ap=angles[i];
    ii=ap.idx[0];
    jj=ap.idx[1];
    kk=ap.idx[2];
    xi=position[ii];
    xj=position[jj];
    xk=position[kk];
    
    drij=real3_subpbc(xi,xj,box);
    drkj=real3_subpbc(xk,xj,box);
    dotp=real3_dot(drij,drkj);
    crop=real3_cross(drij,drkj); // c = a x b
    mcrop=real3_mag(crop);
    t=atan2f(mcrop,dotp);

    // Scaling
    b[0]=0xFFFF & ap.siteBlock[0];
    b[1]=0xFFFF & ap.siteBlock[1];
    if (b[0]) {
      l[0]=lambda[b[0]];
      if (b[1]) {
        l[1]=lambda[b[1]];
      }
    }

    // Interaction
    fangle=ap.kangle*(t-ap.angle0);
    if (b[0] || energy) {
      lEnergy=0.5*ap.kangle*(t-ap.angle0)*(t-ap.angle0);
    }

    // Lambda force
    if (b[0]) {
      realAtomicAdd(&lambdaForce[b[0]],l[1]*lEnergy);
      if (b[1]) {
        realAtomicAdd(&lambdaForce[b[1]],l[0]*lEnergy);
      }
    }

    // Spatial force
    fi=real3_cross(drij,crop);
#warning "division on kernel, was using realRecip before."
    real3_scaleself(&fi, fangle/(mcrop*real3_mag2(drij)));
    at_real3_inc(&force[ii], fi);
    fk=real3_cross(drkj,crop);
    real3_scaleself(&fk,-fangle/(mcrop*real3_mag2(drkj)));
    at_real3_inc(&force[kk], fk);
    fj=real3_add(fi,fk);
    real3_scaleself(&fj,-1);
    at_real3_inc(&force[jj], fj);
  }

  // Energy, if requested
  if (energy) {
    lEnergy*=l[0]*l[1];
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,8);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,16);
    __syncthreads();
    if ((0x1F & threadIdx.x)==0) {
      sEnergy[threadIdx.x>>5]=lEnergy;
    }
    __syncthreads();
    if (threadIdx.x < (blockDim.x>>5)) {
      lEnergy=sEnergy[threadIdx.x];
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    }
    if (threadIdx.x==0) {
      realAtomicAdd(energy,lEnergy);
    }
  }
}

void getforce_angle(System *system,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Msld *m=system->msld;
  int N=p->angleCount;
  int shMem=0;
  real *pEnergy=NULL;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eeangle;
  }

  getforce_angle_kernel<<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->angles_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
}



__device__ void function_torsion(DihePotential dp,real phi,real *fphi,real *lE,bool calcEnergy)
{
  real dphi;

  dphi=dp.ndih*phi-dp.dih0;
  fphi[0]=-dp.kdih*dp.ndih*sinf(dphi);
  if (calcEnergy) {
    lE[0]=dp.kdih*(cosf(dphi)+1);
  }
}

__device__ void function_torsion(ImprPotential ip,real phi,real *fphi,real *lE,bool calcEnergy)
{
  real dphi;

  dphi=phi-ip.imp0;
  dphi-=(2*M_PI)*floor((dphi+M_PI)/(2*M_PI));
  fphi[0]=ip.kimp*dphi;
  if (calcEnergy) {
    lE[0]=0.5*ip.kimp*dphi*dphi;
  }
}


// getforce_dihe_kernel<<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->dihes_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
template <class TorsionPotential>
__global__ void getforce_torsion_kernel(int torsionCount,struct TorsionPotential *torsions,real3 *position,real3 *force,real3 box,real *lambda,real *lambdaForce,real *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int ii,jj,kk,ll;
  TorsionPotential tp;
  real rjk;
  real3 drij,drjk,drkl;
  real3 mvec,nvec;
  real phi,sign,ipr;
  real cosp,sinp;
  real3 dsinp;
  real minv2,ninv2,rjkinv2;
  real p,q;
  real3 fi,fj,fk,fl;
  real ftorsion;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,xj,xk,xl;
  int b[2];
  real l[2]={1,1};

  if (i<torsionCount) {
    // Geometry
    tp=torsions[i];
    ii=tp.idx[0];
    jj=tp.idx[1];
    kk=tp.idx[2];
    ll=tp.idx[3];
    xi=position[ii];
    xj=position[jj];
    xk=position[kk];
    xl=position[ll];

    drij=real3_subpbc(xi,xj,box);
    drjk=real3_subpbc(xj,xk,box);
    drkl=real3_subpbc(xk,xl,box);
    mvec=real3_cross(drij,drjk);
    nvec=real3_cross(drjk,drkl);
    dsinp=real3_cross(mvec,nvec);
    sinp=real3_mag(dsinp);
    cosp=real3_dot(mvec,nvec);
    phi=atan2f(sinp,cosp);
    ipr=real3_dot(drij,nvec);
    sign=(ipr > 0.0) ? -1.0 : 1.0; // Opposite of gromacs because m and n are opposite
    phi=sign*phi;

    // Scaling
    b[0]=0xFFFF & tp.siteBlock[0];
    b[1]=0xFFFF & tp.siteBlock[1];
    if (b[0]) {
      l[0]=lambda[b[0]];
      if (b[1]) {
        l[1]=lambda[b[1]];
      }
    }

    // Interaction
    function_torsion(tp,phi,&ftorsion,&lEnergy, b[0] || energy);

#warning "Division and sqrt in kernel"
    minv2=1/(real3_mag2(mvec));
    ninv2=1/(real3_mag2(nvec));
    rjk=sqrt(real3_mag2(drjk));
    rjkinv2=1/(rjk*rjk);
    fi=real3_scale(-ftorsion*rjk*minv2,mvec);
    at_real3_inc(&force[ii], fi);

    fk=real3_scale(-ftorsion*rjk*ninv2,nvec);
    p=real3_dot(drij,drjk)*rjkinv2;
    q=real3_dot(drkl,drjk)*rjkinv2;
    fj=real3_scale(-p,fi);
    real3_scaleinc(&fj,-q,fk);
    fl=real3_scale(-1,fk);
    at_real3_inc(&force[ll], fl);

    real3_dec(&fk,fj);
    at_real3_inc(&force[kk], fk);

    real3_dec(&fj,fi);
    at_real3_inc(&force[jj], fj);
  }

  // Energy, if requested
  if (energy) {
    lEnergy*=l[0]*l[1];
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,8);
    lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,16);
    __syncthreads();
    if ((0x1F & threadIdx.x)==0) {
      sEnergy[threadIdx.x>>5]=lEnergy;
    }
    __syncthreads();
    if (threadIdx.x < (blockDim.x>>5)) {
      lEnergy=sEnergy[threadIdx.x];
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,1);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,2);
      lEnergy+=__shfl_down_sync(0xFFFFFFFF,lEnergy,4);
    }
    if (threadIdx.x==0) {
      realAtomicAdd(energy,lEnergy);
    }
  }
}

void getforce_dihe(System *system,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Msld *m=system->msld;
  int N=p->diheCount;
  int shMem=0;
  real *pEnergy=NULL;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eedihe;
  }

  getforce_torsion_kernel <DihePotential> <<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->dihes_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
}

void getforce_impr(System *system,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Msld *m=system->msld;
  int N=p->imprCount;
  int shMem=0;
  real *pEnergy=NULL;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eeimpr;
  }

  getforce_torsion_kernel <ImprPotential> <<<(N+BLBO-1)/BLBO,BLBO,shMem,p->bondedStream>>>(N,p->imprs_d,(real3*)s->position_d,(real3*)s->force_d,s->orthBox,m->lambda_d,m->lambdaForce_d,pEnergy);
}
