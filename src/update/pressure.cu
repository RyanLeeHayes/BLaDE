#include "hip/hip_runtime.h"
// #include <nvToolsExt.h>

#include "system/system.h"
#include "run/run.h"
#include "system/state.h"
#include "system/potential.h"
#include "domdec/domdec.h"
#include "rng/rng_cpu.h"
#include "holonomic/rectify.h"



// scale_box_kernel<<<(N+BLUP-1)/BLUP,BLUP,0,system->run->updateStream>>>(N,scaleFactor,system->state->position_d);
__global__ void scale_box_kernel(int N,real_x scaleFactor,real_x *position)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    position[i]*=scaleFactor;
  }
}

__global__ void shift_box_kernel(int N,real3_x shift,real3_x *position)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    position[i].x+=shift.x;
    position[i].y+=shift.y;
    position[i].z+=shift.z;
  }
}

void scale_box(System *system,real_x scaleFactor)
{
  system->state->orthBox.x*=scaleFactor;
  system->state->orthBox.y*=scaleFactor;
  system->state->orthBox.z*=scaleFactor;

  int N=3*system->state->atomCount;
  scale_box_kernel<<<(N+BLUP-1)/BLUP,BLUP,0,system->run->updateStream>>>(N,scaleFactor,(real_x*)system->state->position_d);

  // Nudge the system to remain centered on absolute harmonic restraints
  if (system->potential->harmCount) {
    int N3=system->state->atomCount;
    real3_x shift;
    shift.x=(1-scaleFactor)*system->potential->harmCenter.x;
    shift.y=(1-scaleFactor)*system->potential->harmCenter.y;
    shift.z=(1-scaleFactor)*system->potential->harmCenter.z;
    shift_box_kernel<<<(N3+BLUP-1)/BLUP,BLUP,0,system->run->updateStream>>>(N3,shift,(real3_x*)system->state->position_d);
  }

  // There might be better ways to rectify holonomic constraints after volume update, I just want to avoid having bonds change direction, which will mess with the velocities"
  holonomic_rectify(system);
}

void pressure_coupling(System *system)
{
  State *s=system->state;
  Run *r=system->run;
  Potential *p=system->potential;
  real_e energyOld, energyNew;
  real_x volumeOld, volumeNew;
  real_x scaleFactor;
  real N,kT,dW;

  // hipStreamWaitEvent(r->updateStream,r->forceComplete,0);

  // nvtxRangePushA("pressure_coupling");
  if (system->id==0) {
    // nvtxRangePushA("head node stuff...");
    // Get old energy
    s->recv_energy();
    energyOld=s->energy[eepotential];

    // and print it
    if (system->verbose>0) {
      for (int i=0; i<eeend; i++) {
        fprintf(stdout," %12.4f",s->energy[i]);
      }
      fprintf(stdout,"\n");
    }

    s->backup_position();

    // Change volume
    volumeOld=s->orthBox.x*s->orthBox.y*s->orthBox.z;
    volumeNew=volumeOld+r->volumeFluctuation*system->rngCPU->rand_normal();
    scaleFactor=exp(log(volumeNew/volumeOld)/3);
    scale_box(system,scaleFactor);
    // nvtxRangePop();
  }
  // Call broadcast_box to set orthBox_f, even if only one node
  system->state->broadcast_box(system);

  // Evaluate new energy
  // nvtxRangePushA("update_domdec");
  system->domdec->update_domdec(system,false);
  // nvtxRangePop();
  // nvtxRangePushA("calc_force");
  p->calc_force(0,system); // 0 tells it to calculate energy freqNRG
  // nvtxRangePop();

  if (system->id==0) {
    // nvtxRangePushA("more head node stuff...");
    // Get new energy
    s->recv_energy();
    energyNew=s->energy[eepotential];

    // and print it
    if (system->verbose>0) {
      for (int i=0; i<eeend; i++) {
        fprintf(stdout," %12.4f",s->energy[i]);
      }
      fprintf(stdout,"\n");
    }

    // Compare energy
    N=s->atomCount-(2*p->triangleConsCount+p->branch1ConsCount+2*p->branch2ConsCount+3*p->branch3ConsCount);
    kT=s->leapParms1->kT;
    dW=energyNew-energyOld+system->run->pressure*(volumeNew-volumeOld)-N*kT*log(volumeNew/volumeOld);
    if (system->verbose>0) {
      fprintf(stdout,"dW= %f, dV= %f\n",dW,volumeNew-volumeOld);
    }
    if (system->rngCPU->rand_uniform()<exp(-dW/kT)) { // accept move
      if (system->verbose>0) {
        fprintf(stdout,"Volume move accepted. New volume=%f\n",volumeNew);
      }
    } else {
      if (system->verbose>0) {
        fprintf(stdout,"Volume move rejected. Old volume=%f\n",volumeOld);
      }
      s->restore_position();
    }
    // nvtxRangePop();
  }
  // Call broadcast_box to set orthBox_f, even if only one node
  system->state->broadcast_box(system);
  // nvtxRangePop();
}
