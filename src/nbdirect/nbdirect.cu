#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "system/system.h"
#include "msld/msld.h"
#include "system/state.h"
#include "run/run.h"
#include "system/potential.h"
#include "domdec/domdec.h"
#include "main/defines.h"
#include "main/real3.h"



__global__ void getforce_nbdirect_kernel(int blockCount,int *blockBounds,int *blockPartnerCount,struct DomdecBlockPartners *blockPartners,struct NbondPotential *nbonds,int vdwParameterCount,struct VdwPotential *vdwParameters,struct Cutoffs cutoffs,real3 *position,real3 *force,real *lambda,real *lambdaForce,real *energy)
{
// NYI - maybe energy should be a double
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int iBlock=i/32;
  int jBlock;
  int iCount,jCount;
  int ii,jj;
  int j,jmax;
  int ij,jtmp;
  real r,rinv;
  real3 shift;
  real3 dr;
  NbondPotential inp,jnp;
  real jtmpnp_q;
  real fij,eij;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,xj,xjtmp;
  real3 fi,fj,fjtmp;
  real fli,flj,fljtmp;
  int bi,bj,bjtmp;
  real li,lj,ljtmp,lixljtmp;
  bool testSelf;

  if (iBlock<blockCount) {
    ii=blockBounds[iBlock];
    iCount=blockBounds[iBlock+1]-ii;
    ii+=(i&31);
    if ((i&31)<iCount) {
      inp=nbonds[ii];
      xi=position[ii];
      bi=inp.siteBlock;
      li=1;
      if (bi) li=lambda[0xFFFF & bi];
    }

    fi=make_float3(0,0,0);
    fli=0;

    jmax=blockPartnerCount[iBlock];
    for (j=0; j<jmax; j++) {
      jBlock=blockPartners[j].jBlock;
      shift=blockPartners[j].shift;
      jj=blockBounds[jBlock];
      jCount=blockBounds[jBlock+1]-jj;
      jj+=(i&31);
      if ((i&31)<jCount) {
        jnp=nbonds[jj];
        xj=position[jj];
        real3_inc(&xj,shift);
        bj=jnp.siteBlock;
        lj=1;
        if (bj) lj=lambda[0xFFFF & bj];
      }
      testSelf=(iBlock==jBlock && shift.x==0 && shift.y==0 && shift.z==0);

      fj=make_float3(0,0,0);
      flj=0;

      for (ij=testSelf; ij<32; ij++) {
        jtmp=i+ij;
        jtmpnp_q=__shfl_sync(0xFFFFFFFF,jnp.q,jtmp);
        int jtmpnp_typeIdx=__shfl_sync(0xFFFFFFFF,jnp.typeIdx,jtmp);
        xjtmp.x=__shfl_sync(0xFFFFFFFF,xj.x,jtmp);
        xjtmp.y=__shfl_sync(0xFFFFFFFF,xj.y,jtmp);
        xjtmp.z=__shfl_sync(0xFFFFFFFF,xj.z,jtmp);
        bjtmp=__shfl_sync(0xFFFFFFFF,bj,jtmp);
        ljtmp=__shfl_sync(0xFFFFFFFF,lj,jtmp);

        fjtmp=make_float3(0,0,0);
        fljtmp=0;
        // if ((i&31)<iCount && (jtmp&31)<jCount)
        jtmp=(testSelf?jtmp:(jtmp&31));
        if ((i&31)<iCount && jtmp<jCount) {
          struct VdwPotential vdwp=vdwParameters[inp.typeIdx*vdwParameterCount+jtmpnp_typeIdx];

          // Geometry
          dr=real3_sub(xi,xjtmp);
          // NOTE #warning "Unprotected sqrt"
          r=real3_mag(dr);

          if (r<cutoffs.rCut) {
            rinv=1/r;

            // Scaling
            if ((bi&0xFFFF0000)==(bjtmp&0xFFFF0000)) {
              if (bi==bjtmp) {
                lixljtmp=li;
              } else {
                lixljtmp=0;
              }
            } else {
              lixljtmp=li*ljtmp;
            }

            // interaction
              // Electrostatics
            real br=cutoffs.betaEwald*r;
            real erfcrinv=erfcf(br)*rinv;
            fij=-kELECTRIC*inp.q*jtmpnp_q*(erfcrinv-(2/sqrt(M_PI))*cutoffs.betaEwald*expf(-br*br))*rinv;
            if (bi || bj || energy) {
              eij=kELECTRIC*inp.q*jtmpnp_q*erfcrinv;
            }
              // Van der Waals
            real rinv3=rinv*rinv*rinv;
            real rinv6=rinv3*rinv3;
            // fij+=-(12*(vdwp.c12*rinv6)-6*(vdwp.c6))*rinv6*rinv;
            // if (bi || bj || energy) {
            //   eij+=(vdwp.c12*rinv6-vdwp.c6)*rinv6;
            // }
            // fij*=lixljtmp;
            // See charmm/source/domdec/enbxfast.F90, functions calc_vdw_constants, vdw_attraction, vdw_repulsion
            real rCut3=cutoffs.rCut*cutoffs.rCut*cutoffs.rCut;
            real rSwitch3=cutoffs.rSwitch*cutoffs.rSwitch*cutoffs.rSwitch;
            if (r<cutoffs.rSwitch) {
              fij+=(6*vdwp.c6-12*vdwp.c12*rinv6)*rinv6*rinv;
              if (bi || bj || energy) {
                real dv6=-1/(rCut3*rSwitch3);
                real dv12=-dv6*dv6;
                eij+=vdwp.c12*(rinv6*rinv6-dv12)-vdwp.c6*(rinv6-dv6);
              }
            } else {
              real k6=rCut3/(rCut3-rSwitch3);
              real k12=rCut3*rCut3/(rCut3*rCut3-rSwitch3*rSwitch3);
              real rCutinv3=1/rCut3;
              fij+=(6*vdwp.c6*k6*(rinv3-rCutinv3)*rinv3-12*vdwp.c12*k12*(rinv6-rCutinv3*rCutinv3)*rinv6)*rinv;
              if (bi || bj || energy) {
                eij+=vdwp.c12*k12*(rinv6-rCutinv3*rCutinv3)*(rinv6-rCutinv3*rCutinv3)-vdwp.c6*k6*(rinv3-rCutinv3)*(rinv3-rCutinv3);
              }
            }

            // Lambda force
            if ((bi&0xFFFF0000)==(bjtmp&0xFFFF0000)) {
              if (bi==bjtmp) {
                fli+=eij;
                fljtmp=eij;
              } // No else
            } else {
              fli+=ljtmp*eij;
              fljtmp=li*eij;
            }
        
            // Spatial force
            real3_scaleinc(&fi, fij*rinv,dr);
            fjtmp=real3_scale(-fij*rinv,dr);

            // Energy, if requested
            if (energy) {
              lEnergy+=lixljtmp*eij;
            }
          }
        }
        __syncwarp();
        jtmp=i-ij;
        fj.x+=__shfl_sync(0xFFFFFFFF,fjtmp.x,jtmp);
        fj.y+=__shfl_sync(0xFFFFFFFF,fjtmp.y,jtmp);
        fj.z+=__shfl_sync(0xFFFFFFFF,fjtmp.z,jtmp);
        flj+=__shfl_sync(0xFFFFFFFF,fljtmp,jtmp);
      }
      __syncwarp();
      if ((i&31)<jCount) {
        if (bj) {
          realAtomicAdd(&lambdaForce[0xFFFF & bj],flj);
        }
        at_real3_inc(&force[jj],fj);
      }
    }
    __syncwarp();
    if ((i&31)<iCount) {
      if (bi) {
        realAtomicAdd(&lambdaForce[0xFFFF & bi],fli);
      }
      at_real3_inc(&force[ii],fi);
    }
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

void getforce_nbdirect(System *system,bool calcEnergy)
{
  system->domdec->pack_positions(system);
  system->domdec->cull_blocks(system);

  Potential *p=system->potential;
  State *s=system->state;
  Msld *m=system->msld;
  Domdec *d=system->domdec;
  int N=d->blockCount;
  int shMem=0;
  real *pEnergy=NULL;

  if (calcEnergy) {
    shMem=BLNB*sizeof(real)/32;
    pEnergy=s->energy_d+eenbdirect;
  }

  getforce_nbdirect_kernel<<<(32*N+BLNB-1)/BLNB,BLNB,shMem,p->nbdirectStream>>>(N,d->blockBounds_d,d->blockPartnerCount_d,d->blockPartners_d,d->localNbonds_d,p->vdwParameterCount,p->vdwParameters_d,system->run->cutoffs,d->localPosition_d,d->localForce_d,m->lambda_d,m->lambdaForce_d,pEnergy);
}

