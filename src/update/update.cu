#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#include "system/system.h"
#include "system/state.h"
#include "msld/msld.h"
#include "run/run.h"
#include "system/potential.h"
#include "rng/rng_gpu.h"

#include "main/real3.h"



// See https://aip.scitation.org/doi/abs/10.1063/1.1420460 for barostat
// Nevermind, this barostat is better:
// dx.doi.org/10.1016/j.cplett.2003.12.039
//
// See also for integrator/bond constraints/thermostat
// https://pubs.acs.org/doi/10.1021/jp411770f

// NYI - NPT
// Molecular dynamics simulations of water and biomolecules with a Monte Carlo constant pressure algorithm
// Johan A...vist *, Petra Wennerstro...m, Martin Nervall, Sinisa Bjelic, Bj...rn O. Brandsdal

/*
// Declarations for header
#ifdef CUDAGRAPH
  hipGraph_t updateGraph, updateLambdaGraph;
  hipGraphExec_t updateGraphExec, updateLambdaGraphExec;
#endif
void State::initialize(System *system)
{
#ifdef CUDAGRAPH
  hipStreamBeginCapture(updateStream);
  system->state->rngGPU->rand_normal(2*leapState->N,leapState->random,updateStream);
  update_VO<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,updateStream>>>(*leapState,*leapParms2);
  update_OV<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,updateStream>>>(*leapState,*leapParms2);
  update_R<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,updateStream>>>(*leapState,*leapParms2);
  reset_F<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,updateStream>>>(*leapState);
  hipStreamEndCapture(updateStream,&updateGraph);
  hipGraphInstantiate(&updateGraphExec,updateGraph,NULL,NULL,0);

  hipStreamBeginCapture(updateLambdaStream);
  system->state->rngGPU->rand_normal(2*lambdaLeapState->N,lambdaLeapState->random,updateLambdaStream);
  system->msld->calc_thetaForce_from_lambdaForce(updateLambdaStream);
  update_VO<<<(lambdaLeapState->N+BLUP-1)/BLUP,BLUP,0,updateLambdaStream>>>(*lambdaLeapState,*lambdaLeapParms2);
  update_OV<<<(lambdaLeapState->N+BLUP-1)/BLUP,BLUP,0,updateLambdaStream>>>(*lambdaLeapState,*lambdaLeapParms2);
  update_R<<<(lambdaLeapState->N+BLUP-1)/BLUP,BLUP,0,updateLambdaStream>>>(*lambdaLeapState,*lambdaLeapParms2);
  system->msld->calc_lambda_from_theta(updateLambdaStream);
  reset_F<<<(lambdaLeapState->N+BLUP-1)/BLUP,BLUP,0,updateLambdaStream>>>(*lambdaLeapState);
  hipStreamEndCapture(updateLambdaStream,&updateLambdaGraph);
  hipGraphInstantiate(&updateLambdaGraphExec,updateLambdaGraph,NULL,NULL,0);
#endif
}
// Calling the graph
  hipGraphLaunch(updateGraphExec,r->updateStream);
// Cleaning up the graphs
#ifdef CUDAGRAPH
  hipGraphExecDestroy(updateGraphExec);
  hipGraphDestroy(updateGraph);
#endif
*/

__global__ void update_V(struct LeapState ls,struct LeapParms2 lp1,struct LeapParms2 lp2)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  struct LeapParms2 lp;

  if (i < ls.N1) {
    lp=lp1;
  } else {
    lp=lp2;
  }

  if (i < ls.N) {
    // Force is dU/dx by convention in this program, not -dU/dx
    ls.v[i]=ls.v[i]-lp.fscale*ls.ism[i]*ls.ism[i]*ls.f[i];
  }
}

__global__ void kinetic_energy_kernel(struct LeapState ls,real *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  real lEnergy=0;
  extern __shared__ real sEnergy[];

  if (i<ls.N) {
    if (energy) {
      lEnergy=ls.v[i]/ls.ism[i];
      lEnergy*=0.5*lEnergy;
    }
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

__global__ void update_VROR(struct LeapState ls,struct LeapParms2 lp1,struct LeapParms2 lp2)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  struct LeapParms2 lp;

  if (i < ls.N1) {
    lp=lp1;
  } else {
    lp=lp2;
  }

  if (i < ls.N) {
    // Force is dU/dx by convention in this program, not -dU/dx
    real v=ls.v[i]-lp.fscale*ls.ism[i]*ls.ism[i]*ls.f[i];
    real x=ls.x[i]+lp.fscale*v;
    v=lp.sqrta*v+lp.noise*ls.ism[i]*ls.random[i];
    // Hamiltonian changes here
    v=lp.sqrta*v+lp.noise*ls.ism[i]*ls.random[ls.N+i];
    x=x+lp.fscale*v;
    ls.v[i]=v;
    ls.x[i]=x;
  }
}



void State::update(int step,System *system)
{
  Run *r=system->run;

  hipStreamWaitEvent(r->updateStream,r->forceComplete,0);
  if (system->id==0) {
  // https://pubs.acs.org/doi/10.1021/jp411770f equation 7
  // Use VRORV

  // Resolve lambda forces
  system->msld->calc_thetaForce_from_lambdaForce(r->updateStream,system);
  // Update V from previous step
  update_V<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(*leapState,*leapParms2,*lambdaLeapParms2);
  // Velocity Constraint
  // Kinetic Energy
  if (system->run->step%system->run->freqNRG==0) {
    kinetic_energy_kernel<<<(leapState->N+BLUP-1)/BLUP,BLUP,BLUP*sizeof(real)/32,r->updateStream>>>(*leapState,energy_d+eekinetic);
  }
  // Get Gaussian distributed random numbers
  system->rngGPU->rand_normal(2*leapState->N,leapState->random,r->updateStream);
  // Update VROR
  update_VROR<<<(leapState->N+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(*leapState,*leapParms2,*lambdaLeapParms2);
  // Position Constraint
  // Project lambdas
  system->msld->calc_lambda_from_theta(r->updateStream,system);
  }

  hipEventRecord(r->updateComplete,r->updateStream);
}
