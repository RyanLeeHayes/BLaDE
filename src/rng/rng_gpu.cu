#include "hip/hip_runtime.h"
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>

#include "rng/rng_gpu.h"

void RngGPU::setup()
{
  long long seed=time(NULL);

  // 200 is limit, each of the 200 states can have up to 256 threads
  hipMalloc((void**)&devStates, 200*sizeof(hiprandStateMtgp32_t));
  /* Allocate space for MTGP kernel parameters */
  hipMalloc((void**)&devParams, sizeof(mtgp32_kernel_params_t));
  
  /* Reformat from predefined parameter sets to kernel format, */
  /* and copy kernel parameters to device memory               */
  hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devParams);
  /* Initialize one state per thread block */
  hiprandMakeMTGP32KernelState(devStates, 
    mtgp32dc_params_fast_11213, devParams, 200, seed);
}

__global__ void kernel_normal(hiprandStateMtgp32 *state,int n,real *p)
{
  int i;
  for (i=256*blockIdx.x+threadIdx.x; i<n; i+=200) {
    p[i]=hiprand_normal(&state[blockIdx.x]);
  }
}

__global__ void kernel_uniform(hiprandStateMtgp32 *state,int n,real *p)
{
  int i;
  for (i=256*blockIdx.x+threadIdx.x; i<n; i+=200) {
    p[i]=hiprand_uniform(&state[blockIdx.x]);
  }
}

// Generate n random numbers in the pointer p
void RngGPU::rand_normal(int n,real *p)
{
  int nblocks=(n+256-1)/256;
  nblocks=(nblocks>200)?200:nblocks;
  kernel_normal<<<nblocks,256>>>(devStates,n,p);
}

// Generate n random numbers in the pointer p
void RngGPU::rand_uniform(int n,real *p)
{
  int nblocks=(n+256-1)/256;
  nblocks=(nblocks>200)?200:nblocks;
  kernel_uniform<<<nblocks,256>>>(devStates,n,p);
}
