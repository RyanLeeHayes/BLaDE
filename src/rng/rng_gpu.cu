#include "hip/hip_runtime.h"
// If this is a CHARMM compilation
#ifdef BLADE_IN_CHARMM

#include "rng_gpu.h"
#include <ctime>
#include <cstdio>
#include <cstdlib>

static const char *curandGetErrorString(hiprandStatus_t error)
{
  switch (error)
    {
    case HIPRAND_STATUS_SUCCESS:
      return "HIPRAND_STATUS_SUCCESS";

    case HIPRAND_STATUS_VERSION_MISMATCH:
      return "HIPRAND_STATUS_VERSION_MISMATCH";

    case HIPRAND_STATUS_NOT_INITIALIZED:
      return "HIPRAND_STATUS_NOT_INITIALIZED";

    case HIPRAND_STATUS_ALLOCATION_FAILED:
      return "HIPRAND_STATUS_ALLOCATION_FAILED";

    case HIPRAND_STATUS_TYPE_ERROR:
      return "HIPRAND_STATUS_TYPE_ERROR";

    case HIPRAND_STATUS_OUT_OF_RANGE:
      return "HIPRAND_STATUS_OUT_OF_RANGE";

    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
      return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
      return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

    case HIPRAND_STATUS_LAUNCH_FAILURE:
      return "HIPRAND_STATUS_LAUNCH_FAILURE";

    case HIPRAND_STATUS_PREEXISTING_FAILURE:
      return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case HIPRAND_STATUS_INITIALIZATION_FAILED:
      return "HIPRAND_STATUS_INITIALIZATION_FAILED";

    case HIPRAND_STATUS_ARCH_MISMATCH:
      return "HIPRAND_STATUS_ARCH_MISMATCH";

    case HIPRAND_STATUS_INTERNAL_ERROR:
      return "HIPRAND_STATUS_INTERNAL_ERROR";
    }

  return "<unknown>";
}

#define curandCheck(stmt) do {                                           \
        hiprandStatus_t err = stmt;                                       \
        if (err != HIPRAND_STATUS_SUCCESS) {                              \
	  fprintf(stderr, "Error running %s in file %s, function %s, line %d\n", \
                 #stmt, __FILE__, __FUNCTION__, __LINE__);               \
	  fprintf(stderr, "Error string: %s\n", curandGetErrorString(err)); \
	  exit(1);						         \
        }                                                                \
    } while(0)

RngGPU::RngGPU()
{
  unsigned long long seed = time(NULL);

  /* Create pseudo-random number generator */
  curandCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
  
  /* Set seed */
  curandCheck(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

  rngStream=NULL;
  curandCheck(hiprandSetStream(gen,rngStream));
}

RngGPU::~RngGPU()
{
    curandCheck(hiprandDestroyGenerator(gen));
}

// Generate n random numbers in the pointer p
void RngGPU::rand_normal(int n,real *p,hipStream_t s)
{
  if (rngStream!=s) {
    rngStream=s;
    curandCheck(hiprandSetStream(gen,rngStream));
  }

  curandCheck(hiprandGenerateNormal(gen, p, n, 0.0f, 1.0f));
}

// Generate n random numbers in the pointer p
void RngGPU::rand_uniform(int n,real *p,hipStream_t s)
{
  if (rngStream!=s) {
    rngStream=s;
    curandCheck(hiprandSetStream(gen,rngStream));
  }

  curandCheck(hiprandGenerateUniform(gen, p, n));
}

#else

/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>

#include "rng/rng_gpu.h"

void RngGPU::setup()
{
  long long seed=time(NULL);

  // 200 is limit, each of the 200 states can have up to 256 threads
  hipMalloc((void**)&devStates, 200*sizeof(hiprandStateMtgp32_t));
  /* Allocate space for MTGP kernel parameters */
  hipMalloc((void**)&devParams, sizeof(mtgp32_kernel_params_t));
  
  /* Reformat from predefined parameter sets to kernel format, */
  /* and copy kernel parameters to device memory               */
  hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devParams);
  /* Initialize one state per thread block */
  hiprandMakeMTGP32KernelState(devStates, 
    mtgp32dc_params_fast_11213, devParams, 200, seed);
}

__global__ void kernel_normal(hiprandStateMtgp32 *state,int n,real *p)
{
  int i;
  for (i=256*blockIdx.x+threadIdx.x; i<((n+255)&0xFFFFFF00); i+=200*256) {
    real result=hiprand_normal(&state[blockIdx.x]); // Whole block generates
    if (i<n) p[i]=result; // Only requested threads store it
  }
}

__global__ void kernel_uniform(hiprandStateMtgp32 *state,int n,real *p)
{
  int i;
  for (i=256*blockIdx.x+threadIdx.x; i<((n+255)&0xFFFFFF00); i+=200*256) {
    real result=hiprand_uniform(&state[blockIdx.x]); // Whole block generates
    if (i<n) p[i]=result; // Only requested threads store it
  }
}

// Generate n random numbers in the pointer p
void RngGPU::rand_normal(int n,real *p,hipStream_t s)
{
  int nblocks=(n+256-1)/256;
  nblocks=(nblocks>200)?200:nblocks;
  kernel_normal<<<nblocks,256,0,s>>>(devStates,n,p);
}

// Generate n random numbers in the pointer p
void RngGPU::rand_uniform(int n,real *p,hipStream_t s)
{
  int nblocks=(n+256-1)/256;
  nblocks=(nblocks>200)?200:nblocks;
  kernel_uniform<<<nblocks,256,0,s>>>(devStates,n,p);
}

#endif
