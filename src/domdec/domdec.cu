#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "domdec/domdec.h"
#include "system/system.h"
#include "system/state.h"
#include "run/run.h"
#include "system/potential.h"



Domdec::Domdec()
{
  domain=NULL;
  domain_d=NULL;
  localToGlobal_d=NULL;
  globalToLocal_d=NULL;
  localPosition_d=NULL;
  localForce_d=NULL;
  localNbonds_d=NULL;
  blockSort_d=NULL;
  blockToken_d=NULL;
  blockBounds_d=NULL;
  blockCount=NULL;
  blockCount_d=NULL;
  blockVolume_d=NULL;
  blockCandidateCount_d=NULL;
  blockCandidates_d=NULL;
  blockPartnerCount_d=NULL;
  blockPartners_d=NULL;
  localExcls_d=NULL;
  exclSort_d=NULL;
  sortedExcls_d=NULL;
  blockExcls_d=NULL;
  blockExclCount_d=NULL;
}

Domdec::~Domdec()
{
#warning "No free test"
  free(domain);
  hipFree(domain_d);
  hipFree(localToGlobal_d);
  hipFree(globalToLocal_d);
  hipFree(localPosition_d);
  hipFree(localForce_d);
  hipFree(localNbonds_d);
  hipFree(blockSort_d);
  hipFree(blockToken_d);
  hipFree(blockBounds_d);
  free(blockCount);
  hipFree(blockCount_d);
  hipFree(blockVolume_d);
  hipFree(blockCandidateCount_d);
  hipFree(blockCandidates_d);
  hipFree(blockPartnerCount_d);
  hipFree(blockPartners_d);
  hipFree(localExcls_d);
  hipFree(exclSort_d);
  hipFree(sortedExcls_d);
  hipFree(blockExcls_d);
  hipFree(blockExclCount_d);
}

void Domdec::initialize(System *system)
{
  idDomdec=make_int3(0,0,0);
  if (system->idCount==1) {
    gridDomdec=make_int3(1,1,1);
  } else {
#warning "No 2d or 3d decomposition implemented"
    gridDomdec=make_int3(1,1,system->idCount-1);
    if (system->id!=0) {
      idDomdec=make_int3(0,0,system->id-1);
    }
  }

  int color=(system->idCount==1 || system->id!=0)?0:MPI_UNDEFINED;
  MPI_Comm_split(MPI_COMM_WORLD,color,0,&MPI_COMM_NBOND);

  globalCount=system->state->atomCount;

  // Assume blocks are on average at least 1/3 full, and add some extra blocks for small systems.
  maxBlocks=3*globalCount/32+32;
  real invDensity=(system->state->orthBox.x*system->state->orthBox.y*system->state->orthBox.z)/system->state->atomCount;
  real approxBlockBox=exp(log(32*invDensity)/3);
  real edge=3*approxBlockBox+2*system->run->cutoffs.rCut;
  // edge*edge*edge is the largest volume that can interact with a typically sized box in the worst case. Typically, half these interactions will be taken care of by partner blocks rather than this block, multiplying this expression by 2 means we should have roughly 4 times as many partner spaces as necessary.
  maxPartnersPerBlock=2*((int)(edge*edge*edge/(32*invDensity)));
  fprintf(stdout,"The following parameters are set heuristically at %s:%d, and can cause errors if set too low\n",__FILE__,__LINE__);
  fprintf(stdout,"maxBlocks=%d\n",maxBlocks);
  fprintf(stdout,"maxPartnersPerBlock=%d\n",maxPartnersPerBlock);

  freqDomdec=10;
  // How far two particles, each with hydrogen/unit mass can get in freqDomdec timesteps, if each has 30 kT of kinetic energy. Incredibly rare to violate this.
  cullPad=2*sqrt(30*kB*system->run->T/1)*freqDomdec*system->run->dt;
  maxBlockExclCount=4*system->potential->exclCount+1024; // only 32*exclCount is guaranteed
  fprintf(stdout,"freqDomdec=%d (how many steps before domain reset)\n",freqDomdec);
  fprintf(stdout,"cullPad=%g (spatial padding for considering which blocks could interact\n",cullPad);
  fprintf(stdout,"maxBlockExclCount=%d\n",maxBlockExclCount);

  domain=(int*)calloc(globalCount,sizeof(int));
  hipMalloc(&domain_d,globalCount*sizeof(int));
  hipMalloc(&localToGlobal_d,globalCount*sizeof(int));
  hipMalloc(&globalToLocal_d,globalCount*sizeof(int));
  hipMalloc(&localPosition_d,globalCount*sizeof(real3));
  hipMalloc(&localForce_d,globalCount*sizeof(real3));
  hipMalloc(&localNbonds_d,globalCount*sizeof(struct NbondPotential));
  hipMalloc(&blockSort_d,(globalCount+1)*sizeof(struct DomdecBlockSort));
  hipMalloc(&blockToken_d,(globalCount+1)*sizeof(struct DomdecBlockToken));
  hipMalloc(&blockBounds_d,maxBlocks*sizeof(int));
  blockCount=(int*)calloc(gridDomdec.x*gridDomdec.y*gridDomdec.z+1,sizeof(int));
  hipMalloc(&blockCount_d,(gridDomdec.x*gridDomdec.y*gridDomdec.z+1)*sizeof(int));
  hipMalloc(&blockVolume_d,maxBlocks*sizeof(struct DomdecBlockVolume));
  hipMalloc(&blockCandidateCount_d,maxBlocks*sizeof(int));
  hipMalloc(&blockCandidates_d,maxBlocks*maxPartnersPerBlock*sizeof(struct DomdecBlockPartners));
  hipMalloc(&blockPartnerCount_d,maxBlocks*sizeof(int));
  hipMalloc(&blockPartners_d,maxBlocks*maxPartnersPerBlock*sizeof(struct DomdecBlockPartners));

  hipMalloc(&localExcls_d,(system->potential->exclCount+1)*sizeof(struct ExclPotential));
  hipMalloc(&exclSort_d,(system->potential->exclCount+1)*sizeof(struct DomdecBlockSort));
  hipMalloc(&sortedExcls_d,system->potential->exclCount*sizeof(struct ExclPotential));
  hipMalloc(&blockExcls_d,maxBlockExclCount*sizeof(int));
  hipMalloc(&blockExclCount_d,sizeof(int));

  reset_domdec(system);
}

void Domdec::reset_domdec(System *system)
{
  // Puts each atom in a specific domain/box controlled by one GPU
  assign_domain(system);
  // Splits domains into blocks, or groups of up to 32 nearby atoms
  assign_blocks(system);
  // Cull blocks to get a candidate list
  cull_blocks(system);
  // Sets up exclusion data structures
  setup_exclusions(system);
}

void Domdec::update_domdec(System *system,bool resetFlag)
{
  if (resetFlag) {
    system->domdec->reset_domdec(system);
  } else if (system->idCount>1) {
    system->state->broadcast_position(system);
  }
}
