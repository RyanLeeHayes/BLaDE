#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "domdec/domdec.h"
#include "io/io.h"
#include "system/system.h"
#include "system/state.h"
#include "system/potential.h"
#include "run/run.h"
#include "main/real3.h"



__host__ __device__ static inline
bool operator<(const DomdecBlockToken& a,const DomdecBlockToken& b)
{
  return (a.domain<b.domain || (a.domain==b.domain &&
         (a.ix<b.ix || (a.ix==b.ix &&
         (a.iy<b.iy || (a.iy==b.iy &&
         (a.z<b.z)))))));
}

// assign_blocks_get_tokens_kernel<<<(globalCount+BLUP-1)/BLUP,BLUP,0,system->update->updateStream>>>(globalCount,idDomdec,gridDomdec,domainDiv,domain_d,system->state->position_d,system->state->orthBox,blockToken_d);
__global__ void assign_blocks_get_tokens_kernel(int globalCount,int3 gridDomdec,int2 domainDiv,int *domain,real3 *position,real3 box,struct DomdecBlockToken *tokens)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int3 idDomdec;
  real3 xi;
  real posInDomain;
  struct DomdecBlockToken token;

  if (i<globalCount+1) {
    if (i==globalCount) {
      token.domain=-1;
    } else {
      token.domain=domain[i];
      idDomdec.x=token.domain/(gridDomdec.y*gridDomdec.z);
      idDomdec.y=token.domain/gridDomdec.z-idDomdec.x*gridDomdec.y;
      xi=position[i];
      posInDomain=xi.x*gridDomdec.x/box.x-idDomdec.x;
      token.ix=(int)floor(posInDomain*domainDiv.x);
      // token.ix-=(token.ix>=domainDiv.x?domainDiv.x:0);
      // token.ix+=(token.ix<0?domainDiv.x:0);
      // No need to wrap, already in box. Fudge it in for rounding errors
      token.ix=(token.ix>=domainDiv.x?(domainDiv.x-1):token.ix);
      token.ix=(token.ix<0?0:token.ix);

      posInDomain=xi.y*gridDomdec.y/box.y-idDomdec.y;
      token.iy=(int)floor(posInDomain*domainDiv.y);
      // token.iy-=(token.iy>=domainDiv.y?domainDiv.y:0);
      // token.iy+=(token.iy<0?domainDiv.y:0);
      // No need to wrap, already in box. Fudge it in for rounding errors
      token.iy=(token.iy>=domainDiv.y?(domainDiv.y-1):token.iy);
      token.iy=(token.iy<0?0:token.iy);

      token.z=xi.z;
    }
    tokens[i]=token;
  }
}

__global__ void assign_blocks_grow_tree_kernel(int globalCount,struct DomdecBlockToken *tokens,DomdecBlockSort *sort)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int leafPos=globalCount; // Root is at end of array
  int nextLeafPos, *nextLeafPosPointer;
  struct DomdecBlockToken token, leafPosToken;
  bool placed=false;

  if (i<globalCount) {
    token=tokens[i];
    if (token.ix>=0) {
      while (!placed) {
        leafPosToken=tokens[leafPos];
        if (leafPosToken<token) {
          nextLeafPosPointer=&sort[leafPos].upper;
        } else {
          nextLeafPosPointer=&sort[leafPos].lower;
        }
        nextLeafPos=*nextLeafPosPointer;
        if (nextLeafPos==-1) { // Try to plant leaf here
          nextLeafPos=atomicCAS(nextLeafPosPointer,-1,i);
          if (nextLeafPos==-1) { // Planting was successful
            placed=true;
            sort[i].root=leafPos;
          }
        }
        if (!placed) {
          leafPos=nextLeafPos;
        }
      }
    }
  }
}

// Work back up the tree to the root, counting leaves
__global__ void assign_blocks_count_tree_kernel(int globalCount,struct DomdecBlockToken *tokens,volatile DomdecBlockSort *sort)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int leafPos, nextLeafPos;
  struct DomdecBlockSort s;
  int count;
  int whoAmI;
  int findWhoAmI;
  bool sister; // boolean for whether a sister exists at a particular level
  bool finished=false;

  if (i<globalCount) {
    s=((struct DomdecBlockSort*)sort)[i];
    // If this is a terminal leaf, start counting up the tree
    if (s.root!=-1 && s.lower==-1 && s.upper==-1) {
      sort[i].lowerCount=0;
      sort[i].upperCount=0;
      leafPos=i;
      while (!finished) {
        nextLeafPos=sort[leafPos].root;
        count=sort[leafPos].lowerCount+sort[leafPos].upperCount+1;
        sister=true;
        
        findWhoAmI=sort[nextLeafPos].lower;
        if (findWhoAmI==-1) {
          sort[nextLeafPos].lowerCount=0;
          sister=false;
        } else if (findWhoAmI==leafPos) {
          sort[nextLeafPos].lowerCount=count;
          whoAmI=0;
        }
        findWhoAmI=sort[nextLeafPos].upper;
        if (findWhoAmI==-1) {
          sort[nextLeafPos].upperCount=0;
          sister=false;
        } else if (findWhoAmI==leafPos) {
          sort[nextLeafPos].upperCount=count;
          whoAmI=1;
        }

        // Try to tell sister to go up tree
        if (sister) {
          if (atomicCAS((int*)&sort[nextLeafPos].whoCounts,-1,1-whoAmI)==-1) { // Succeeded
            finished=true;
          }
        }
          
        if (nextLeafPos==globalCount) { // Made it all the way up the tree
          finished=true;
        }

        leafPos=nextLeafPos;
      }
    }
  }
}

__global__ void assign_blocks_localToGlobal_kernel(int globalCount,struct DomdecBlockSort *sort,int *localToGlobal)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int leafPos=globalCount;
  int count=0;
  int nextCount;
  bool finished=false;

  if (i<globalCount) {
    // Move past root, which doesn't represent an actual atom
    leafPos=sort[leafPos].upper;
    while (!finished) {
      nextCount=count+sort[leafPos].lowerCount;
      if (nextCount<i) {
        leafPos=sort[leafPos].upper;
        count=nextCount+1;
      } else if (nextCount>i) {
        leafPos=sort[leafPos].lower;
      } else {
        finished=true;
      }
    }
    localToGlobal[i]=leafPos;
  }
}

// assign_blocks_blockBounds_kernel<<<1,idCount*domainDiv.x*domainDiv.y+1,2*(idCount*domainDiv.x*domainDiv.y+1)*sizeof(int),system->update->updateStream>>>(domainDiv,globalCount,localToGlobal_d,blockToken_d,blockCount_d,blockBounds_d);
// Input
// domainDiv - how many blocks a domain is divided into in the x and y directions
// localCount - entries in localToGlobal
// localToGlobal - list for binary search
// tokens - tokens that were used for making the tree structure. Contain information on which column a particle is in
// Output
// blockCount - pointer to a single int for the total number of blocks
// blockBounds - indices (in the local indexing) of first atom in each block
__global__ void assign_blocks_blockBounds_kernel(int domainCount,int2 domainDiv,int globalCount,int *localToGlobal,struct DomdecBlockToken *tokens,int *blockCount,int *blockBounds,int maxBlocks)
{
  int i,i0;
  int domain;
  int ix,iy;
  int probePos,hwidth,j;
  int blocksInColumn;
  extern __shared__ int columnBounds[]; // Two shared arrays of size blockDim.x+1
  int *cumBlocks=columnBounds+blockDim.x+1; // Two shared arrays of size blockDim.x+1
  struct DomdecBlockToken token,probeToken;

  if (threadIdx.x==0) {
    blockCount[0]=0;
    columnBounds[0]=0;
    cumBlocks[0]=0;
  }

  __syncthreads();

  // Loop through domains (slabs in z direction)
  for (domain=0; domain<domainCount; domain++) {
    // Loops through columns in x and y dimension
    for (i0=0; i0<domainDiv.x*domainDiv.y; i0+=blockDim.x) {
      i=i0+threadIdx.x; // column index
      ix=i/domainDiv.y; // x column index
      iy=i-ix*domainDiv.y; // y column index

      // Create search token
      token.domain=domain;
      token.ix=ix;
      token.iy=iy;
      token.z=INFINITY; // Infinity finds first element in next column

      // Set bounds for binary search
      int lowerPos=-1;
      int upperPos=globalCount;

      // Find half of next highest power of 2 above localCount+1
      hwidth=globalCount; // (localCount+1)-1
      hwidth|=hwidth>>1;
      hwidth|=hwidth>>2;
      hwidth|=hwidth>>4;
      hwidth|=hwidth>>8;
      hwidth|=hwidth>>16;
      hwidth++;
      hwidth=hwidth>>1;

      // Do binary search
      for (; hwidth>0; hwidth=hwidth>>1) {
        probePos=lowerPos+hwidth;
        if (probePos<upperPos) {
          probeToken=tokens[localToGlobal[probePos]];
          if (probeToken<token) {
            lowerPos=probePos;
          } else {
            upperPos=probePos;
          }
        }
      }

      // Search complete, save to shared array columnBounds
      int is0=threadIdx.x; // shared memory i position
      int is1=threadIdx.x+1; // shared memory i+1 position
      columnBounds[is1]=upperPos;

      __syncthreads();

      lowerPos=columnBounds[is0];
      blocksInColumn=upperPos-lowerPos;
      blocksInColumn=(blocksInColumn+31)/32;

      // Compute cumulative sum of blocks in all previous columns in shared array cumBlocks
      // Requires shared memory because the number of columns is unrelated to warp size
      cumBlocks[is1]=blocksInColumn;
      for (hwidth=1; hwidth<blockDim.x+1; hwidth*=2) {
        __syncthreads();
        if (hwidth&is1) {
          cumBlocks[is1]+=cumBlocks[(is1|(hwidth-1))-hwidth];
        }
      }

      __syncthreads();

      // Save the block bounds into the blockBounds array
      if (i<domainDiv.x*domainDiv.y) {
        int j0=cumBlocks[is0];
        if (j0+blocksInColumn<=maxBlocks) {
        for (j=0; j<blocksInColumn; j++) {
          blockBounds[j+j0]=lowerPos+32*j;
        }
        } else if (j0<maxBlocks) {
// #warning "printf in kernel, this doesn't affect occupancy of 93.8\% on 2080 TI."
          printf("Error: Overflow of maxBlocks. Use \"run setvariable domdecheuristic off\" - except that reallocation is not implemented here\n");
        }
      }

      __syncthreads();

      // Update columnBounds and cumBlocks for next part of this domain
      if (threadIdx.x==0) {
        columnBounds[0]=columnBounds[blockDim.x];
        cumBlocks[0]=cumBlocks[blockDim.x];
      }
      __syncthreads();
    }

    // Update blockCount for next domain after this one is complete
    if (threadIdx.x==0) {
      blockCount[domain+1]=cumBlocks[0];
    }
    __syncthreads();
  }

  // Set final bound
  if (threadIdx.x==0) {
    // blockBounds[blockCount[domainCount]]=globalCount;
    blockBounds[cumBlocks[0]]=columnBounds[0]; // Equivalent statements
  }
}

__global__ void assign_blocks_localNbonds_kernel(int blockCount,int *blockBounds,int *localToGlobal,int *globalToLocal,NbondPotential *nbonds,NbondPotential *localNbonds)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int iBlock=i/32;
  int iLocal,iGlobal,atomsInBlock;

  if (iBlock<blockCount) {
    iLocal=blockBounds[iBlock];
    atomsInBlock=blockBounds[iBlock+1]-iLocal;
    iLocal+=(i&31);
    if ((i&31)<atomsInBlock) {
      iGlobal=localToGlobal[iLocal];
      globalToLocal[iGlobal]=iLocal;
      localNbonds[i]=nbonds[iGlobal];
    }
  }
}

/* OLD __global__ void assign_blocks_finish_local_kernel(int globalCount,int *localToGlobal,real3 *position,real3 *localPosition)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<globalCount) {
    int iGlobal=localToGlobal[i];
    localPosition[i]=position[iGlobal];
  }
}*/

//    assign_blocks_finish_local_kernel<<<(32*blockCount+BLUP-1)/BLUP,BLUP,0,system->update->updateStream>>>(blockCount,blockBounds_d,localToGlobal_d,(real3*)system->state->position_d,localPosition_d,blockVolume_d);
__global__ void assign_blocks_localPosition_kernel(int blockCount,int *blockBounds,int *localToGlobal,real3 *position,real3 *localPosition,struct DomdecBlockVolume *blockVolume)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int iBlock=i/32;
  int j,iLocal,iGlobal,atomsInBlock;
  real3 xi;
  struct DomdecBlockVolume v,vCompare;

  if (iBlock<blockCount) {
    // Copy over positions to local structure
    iLocal=blockBounds[iBlock];
    atomsInBlock=blockBounds[iBlock+1]-iLocal;
    iLocal+=(i&31);
    if ((i&31)<atomsInBlock) {
      iGlobal=localToGlobal[iLocal];
      xi=position[iGlobal];
      localPosition[i]=xi;
    }

    // Find extreme values
    v.max=xi;
    v.min=xi;
    for (j=1; j<32; j*=2) {
      vCompare.min.x=__shfl_down_sync(0xFFFFFFFF,v.min.x,j);
      vCompare.min.y=__shfl_down_sync(0xFFFFFFFF,v.min.y,j);
      vCompare.min.z=__shfl_down_sync(0xFFFFFFFF,v.min.z,j);
      vCompare.max.x=__shfl_down_sync(0xFFFFFFFF,v.max.x,j);
      vCompare.max.y=__shfl_down_sync(0xFFFFFFFF,v.max.y,j);
      vCompare.max.z=__shfl_down_sync(0xFFFFFFFF,v.max.z,j);
      if ((i&31)+j<atomsInBlock) {
        v.min.x=(v.min.x<vCompare.min.x?v.min.x:vCompare.min.x);
        v.min.y=(v.min.y<vCompare.min.y?v.min.y:vCompare.min.y);
        v.min.z=(v.min.z<vCompare.min.z?v.min.z:vCompare.min.z);
        v.max.x=(v.max.x>vCompare.max.x?v.max.x:vCompare.max.x);
        v.max.y=(v.max.y>vCompare.max.y?v.max.y:vCompare.max.y);
        v.max.z=(v.max.z>vCompare.max.z?v.max.z:vCompare.max.z);
      }
    }
    if ((i&31)==0) {
      blockVolume[iBlock]=v;
    }
  }
}

void Domdec::assign_blocks(System *system)
{
  Run *r=system->run;

  if (id>=0) { 

    // Get the tokens for sorting

    real3 box;
    if (system->state->typeBox) {
      box.x=system->state->tricBox_f.a.x;
      box.y=system->state->tricBox_f.b.y;
      box.z=system->state->tricBox_f.c.z;
    } else {
      box=system->state->orthBox_f;
    }

    assign_blocks_get_tokens_kernel<<<(globalCount+1+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(globalCount,gridDomdec,domainDiv,domain_d,(real3*)system->state->position_fd,box,blockToken_d);

    // Make the tree structure

    hipMemsetAsync(blockSort_d,-1,(globalCount+1)*sizeof(struct DomdecBlockSort),r->updateStream);

    assign_blocks_grow_tree_kernel<<<(globalCount+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(globalCount,blockToken_d,blockSort_d);
    assign_blocks_count_tree_kernel<<<(globalCount+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(globalCount,blockToken_d,blockSort_d);

    // Create sorted structures

    assign_blocks_localToGlobal_kernel<<<(globalCount+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(globalCount,blockSort_d,localToGlobal_d);

    int ndiv=domainDiv.x*domainDiv.y;
    ndiv=((ndiv>1024)?1024:ndiv);
    assign_blocks_blockBounds_kernel<<<1,ndiv,2*(ndiv+1)*sizeof(int),r->updateStream>>>(idCount,domainDiv,globalCount,localToGlobal_d,blockToken_d,blockCount_d,blockBounds_d,maxBlocks);

    hipMemcpy(blockCount,blockCount_d,(idCount+1)*sizeof(int),hipMemcpyDeviceToHost);

    assign_blocks_localNbonds_kernel<<<(32*blockCount[idCount]+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(blockCount[idCount],blockBounds_d,localToGlobal_d,globalToLocal_d,system->potential->nbonds_d,localNbonds_d);

    // Redundant with pack_positions, needed for call to cull
    assign_blocks_localPosition_kernel<<<(32*blockCount[idCount]+BLUP-1)/BLUP,BLUP,0,r->updateStream>>>(blockCount[idCount],blockBounds_d,localToGlobal_d,(real3*)system->state->position_fd,localPosition_d,blockVolume_d);
  }
}

void Domdec::pack_positions(System *system)
{
  Run *r=system->run;
  int N=blockCount[idCount];
  if (id>=0) {
    assign_blocks_localPosition_kernel<<<(32*N+BLUP-1)/BLUP,BLUP,0,r->nbdirectStream>>>(N,blockBounds_d,localToGlobal_d,(real3*)system->state->position_fd,localPosition_d,blockVolume_d);
  }
}

__global__ void unpack_forces_kernel(int blockCount,int *blockBounds,int *localToGlobal,real3_f *force,real3_f *localForce)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int iBlock=i/32;
  int iLocal,atomsInBlock;

  if (iBlock<blockCount) {
    iLocal=blockBounds[iBlock];
    atomsInBlock=blockBounds[iBlock+1]-iLocal;
    iLocal+=(i&31);
    if ((i&31)<atomsInBlock) {
      at_real3_inc(&force[localToGlobal[iLocal]],localForce[i]);
    }
  }
}

void Domdec::unpack_forces(System *system)
{
  Run *r=system->run;
  int N=blockCount[idCount];
  if (id>=0) {
    unpack_forces_kernel<<<(32*N+BLUP-1)/BLUP,BLUP,0,r->nbdirectStream>>>(N,blockBounds_d,localToGlobal_d,(real3_f*)system->state->force_d,localForce_d);
  }
}
