#include "hip/hip_runtime.h"

#include "atoms.h"
#include "leapparms.h"
#include "mersenne.h"

#include "defines.h"

#include <stdlib.h>


struct_atoms* alloc_atoms(int N,int Ni,int Nf,struct_leapparms lp,hipStream_t strm,int pbcflag)
{
  struct_atoms *atoms;

  atoms=(struct_atoms*) malloc(sizeof(struct_atoms));

  atoms->N=N;
  atoms->Ni=Ni;
  atoms->Nf=Nf;
  hipMalloc(&(atoms->x),N*sizeof(real));
  hipMalloc(&(atoms->f),N*sizeof(real));
  hipMalloc(&(atoms->v),N*sizeof(real));
  hipMalloc(&(atoms->vhalf),N*sizeof(real));
  hipMalloc(&(atoms->m),N*sizeof(real));
  hipMalloc(&(atoms->misqrt),N*sizeof(real));
  hipMalloc(&(atoms->Vs_delay),N*sizeof(real));

  if (pbcflag) {
    hipMalloc(&(atoms->fixx),N*sizeof(unsigned fixreal));
    hipMalloc(&(atoms->fixreal2real),DIM3*sizeof(real));
    hipMalloc(&(atoms->real2fixreal),DIM3*sizeof(real));
  } else {
    atoms->fixx=NULL;
    atoms->fixreal2real=NULL;
    atoms->real2fixreal=NULL;
  }

  atoms->lp=lp;
  // alloc_mtstate(atoms->mts,strm,time(NULL),DIM3*N);
  alloc_mtstate(atoms->mts,strm,time(NULL),N); // already multiplied by 3

  return atoms;
}


void free_atoms(struct_atoms* atoms)
{
  hipFree(atoms->x);
  hipFree(atoms->f);
  hipFree(atoms->v);
  hipFree(atoms->vhalf);
  hipFree(atoms->m);
  hipFree(atoms->misqrt);
  hipFree(atoms->Vs_delay);

  if (atoms->fixx) {
    hipFree(atoms->fixx);
    hipFree(atoms->fixreal2real);
    hipFree(atoms->real2fixreal);
  }

  free_mtstate(atoms->mts,atoms->N);

  free(atoms);
}


__global__ void setfloat2fix(struct_atoms at,real* box)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if (i<DIM3) {
    at.real2fixreal[i]=FREAL_MAX/box[i];
    at.fixreal2real[i]=box[i]/FREAL_MAX;
  }
}


__global__ void float2fix(struct_atoms at)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  real fixx;

  if (i<at.N) {
    fixx=at.x[i]*at.real2fixreal[i%DIM3];
    at.fixx[i]=(unsignedfixreal) (fixx-FREAL_MAX*floor(fixx/FREAL_MAX));
  }
}


__global__ void fix2float(struct_atoms at)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if (i<at.N) {
    at.x[i]=at.fixx[i]*at.fixreal2real[i%DIM3];
  }
}
