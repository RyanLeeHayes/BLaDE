#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "restrain.h"
#include "main/defines.h"
#include "system/system.h"
#include "system/state.h"
#include "run/run.h"
#include "system/potential.h"

#include "main/real3.h"

template <bool flagBox,typename box_type>
__global__ void getforce_noe_kernel(int noeCount,struct NoePotential *noes,real3 *position,real3_f *force,box_type box,real_e *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  NoePotential noep;
  real r,r_r0;
  real3 dr;
  real fnoe=0;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,xj;
  
  if (i<noeCount) {
    // Geometry
    noep=noes[i];
    xi=position[noep.i];
    xj=position[noep.j];
    dr=real3_subpbc<flagBox>(xi,xj,box);
    r=real3_mag<real>(dr);
    if (r<noep.rmin) {
      r_r0=r-noep.rmin;
      fnoe=noep.kmin*r_r0;
      if (energy) lEnergy=((real)0.5)*fnoe*r_r0;
    } else if (r>noep.rmax) {
      r_r0=r-noep.rmax;
      if (noep.rswitch>0 && r_r0>noep.rswitch) {
        real bswitch=(noep.rpeak-noep.rswitch)/noep.nswitch*pow(noep.rswitch,noep.nswitch+1);
        real aswitch=0.5*noep.rswitch*noep.rswitch-noep.rpeak*noep.rswitch-noep.rswitch*(noep.rpeak-noep.rswitch)/noep.nswitch;
        fnoe=noep.kmax*(noep.rpeak-bswitch*pow(r_r0,-noep.nswitch-1));
        if (energy) lEnergy=noep.kmax*(aswitch+bswitch*pow(r_r0,-noep.nswitch)+noep.rpeak*r_r0);
      } else {
        fnoe=noep.kmax*r_r0;
        if (energy) lEnergy=((real)0.5)*fnoe*r_r0;
      }
    }
    // Spatial force
    at_real3_scaleinc(&force[noep.i], fnoe/r,dr);
    at_real3_scaleinc(&force[noep.j],-fnoe/r,dr);
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

template <bool flagBox,typename box_type>
void getforce_noeT(System *system,box_type box,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Run *r=system->run;
  int N;
  int shMem=0;
  real_e *pEnergy=NULL;

  if (r->calcTermFlag[eebias]==false) return;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebias;
  }

  N=p->noeCount;
  if (N>0) getforce_noe_kernel<flagBox><<<(N+BLBO-1)/BLBO,BLBO,shMem,r->biaspotStream>>>(N,p->noes_d,(real3*)s->position_fd,(real3_f*)s->force_d,box,pEnergy);
}

void getforce_noe(System *system,bool calcEnergy)
{
  if (system->state->typeBox) {
    getforce_noeT<true>(system,system->state->tricBox_f,calcEnergy);
  } else {
    getforce_noeT<false>(system,system->state->orthBox_f,calcEnergy);
  }
}

template <bool flagBox,typename box_type>
__global__ void getforce_harm_kernel(int harmCount,struct HarmonicPotential *harms,real3 *position,real3_f *force,box_type box,real_e *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int ii;
  real r2;
  real3 dr;
  HarmonicPotential hp;
  real krnm2;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,x0;
  
  if (i<harmCount) {
    // Geometry
    hp=harms[i];
    ii=hp.idx;
    xi=position[ii];
    x0=hp.r0;
// NOTE #warning "Unprotected division"
    dr=real3_subpbc<flagBox>(xi,x0,box);
    r2=real3_mag2<real>(dr);
    krnm2=(r2 ? (hp.k*pow(r2,((real)0.5)*hp.n-1)) : 0); // NaN guard it
    
    if (energy) {
      lEnergy=krnm2*r2;
    }
    at_real3_scaleinc(&force[ii], hp.n*krnm2,dr);
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

template <bool flagBox,typename box_type>
void getforce_harmT(System *system,box_type box,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Run *r=system->run;
  int N;
  int shMem=0;
  real_e *pEnergy=NULL;

  if (r->calcTermFlag[eebias]==false) return;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebias;
  }

  N=p->harmCount;
  if (N>0) getforce_harm_kernel<flagBox><<<(N+BLBO-1)/BLBO,BLBO,shMem,r->biaspotStream>>>(N,p->harms_d,(real3*)s->position_fd,(real3_f*)s->force_d,box,pEnergy);
}

void getforce_harm(System *system,bool calcEnergy)
{
  if (system->state->typeBox) {
    getforce_harmT<true>(system,system->state->tricBox_f,calcEnergy);
  } else {
    getforce_harmT<false>(system,system->state->orthBox_f,calcEnergy);
  }
}
